#include "hip/hip_runtime.h"
#include "pathtracer.h"
#include "camera.h"
#include "scene.h"
#include "bvh.h"
#include ""

Camera* dev_camera;
int maxDepth;
float3* dev_image, *dev_color;
LinearBVHNode* dev_bvh_nodes;
Primitive* dev_primitives;
Material* dev_materials;
Area* dev_lights;
float* dev_light_distribution;
float4* hdr_map;
uchar4** dev_textures;
int* texture_size;//0 1Ϊ��һ��ͼ�ĳ��� 2 3Ϊ�ڶ���ͼ�ĳ����Դ�����
texture<float4, 1, hipReadModeElementType> hdr_texture;

__device__ Camera* kernel_camera;
__device__ int kernel_hdr_width, kernel_hdr_height;
__device__ float3* kernel_acc_image, *kernel_color;
__device__ LinearBVHNode* kernel_linear;
__device__ Primitive* kernel_primitives;
__device__ Material* kernel_materials;
__device__ Area* kernel_lights;
__device__ uchar4** kernel_textures;
__device__ int* kernel_texture_size;
__device__ float* kernel_light_distribution;
__device__ int kernel_light_distribution_size;
__device__ bool kernel_hdr_isvalid;
//��ͬ������Ҫ��ͬ��epsilon����֪����ô�����ŵ�ʵ��
__device__ float kernel_epsilon;

__device__ inline unsigned int WangHash(unsigned int seed)
{
	seed = (seed ^ 61) ^ (seed >> 16);
	seed = seed + (seed << 3);
	seed = seed ^ (seed >> 4);
	seed = seed * 0x27d4eb2d;
	seed = seed ^ (seed >> 15);

	return seed;
}

__device__ inline float DielectricFresnel(float cosi, float cost, const float& etai, const float& etat){
	float Rparl = (etat * cosi - etai * cost) / (etat * cosi + etai * cost);
	float Rperp = (etai * cosi - etat * cost) / (etai * cosi + etat * cost);

	return (Rparl * Rparl + Rperp * Rperp) * 0.5f;
}

__device__ inline float3 ConductFresnel(float cosi, const float3& eta, const float3& k){
	float3 tmp = (eta * eta + k * k) * cosi * cosi;
	float3 Rparl2 = (tmp - eta * cosi * 2.f + 1.f) /
		(tmp + eta * cosi * 2.f + 1.f);
	float3 tmp_f = (eta * eta + k * k);
	float3 Rperp2 = (tmp_f - eta * cosi * 2.f + cosi * cosi) /
		(tmp_f + eta * cosi * 2.f + cosi * cosi);
	return (Rparl2 + Rperp2) * 0.5f;
}

__device__ inline float GGX_D(float3& wh, float3& normal, float3 dpdu, float alphaU, float alphaV){
	float costheta = dot(wh, normal);
	if (costheta <= 0.f) return 0.f;
	costheta = clamp(costheta, 0.f, 1.f);
	float costheta2 = costheta*costheta;
	float sintheta2 = 1.f - costheta2;
	float costheta4 = costheta2*costheta2;
	float tantheta2 = sintheta2 / costheta2;

	float3 uu = dpdu;
	float3 dir = normalize(wh - costheta*normal);
	float cosphi = dot(dir, uu);
	float cosphi2 = cosphi*cosphi;
	float sinphi2 = 1.f - cosphi2;
	float sqrD = 1.f + tantheta2*(cosphi2 / (alphaU*alphaU) + sinphi2 / (alphaV*alphaV));
	return 1.f / (PI*alphaU*alphaV*costheta4*sqrD*sqrD);
}

__device__ inline float SmithG(float3& w, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV){
	float wdn = dot(w, normal);
	if (wdn * dot(w, wh) < 0.f)	return 0.f;
	float sintheta = sqrtf(clamp(1.f - wdn*wdn, 0.f, 1.f));
	float tantheta = sintheta / wdn;
	if (isinf(tantheta)) return 0.f;

	float3 uu = dpdu;
	float3 dir = normalize(w - wdn*normal);
	float cosphi = dot(dir, uu);
	float cosphi2 = cosphi*cosphi;
	float sinphi2 = 1.f - cosphi2;
	float alpha2 = cosphi2 * (alphaU*alphaU) + sinphi2 * (alphaV*alphaV);
	float sqrD = alpha2*tantheta*tantheta;
	return 2.f / (1.f + sqrtf(1 + sqrD));
}

__device__ inline float GGX_G(float3& wo, float3& wi, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV){
	return SmithG(wo, normal, wh, dpdu, alphaU, alphaV)*SmithG(wi, normal, wh, dpdu, alphaU, alphaV);
}

__device__ inline float3 SampleGGX(float alphaU, float alphaV, float u1, float u2){
	if (alphaU == alphaV){
		float costheta = sqrtf((1.f - u1) / (u1*(alphaU*alphaV - 1.f) + 1.f));
		float sintheta = sqrtf(1.f - costheta*costheta);
		float phi = 2 * PI*u2;
		float cosphi = cosf(phi);
		float sinphi = sinf(phi);

		return{
			sintheta*cosphi,
			costheta,
			sintheta*sinphi
		};
	}
	else{
		float phi;
		if (u2 <= 0.25) phi = atan(alphaV / alphaU*tan(TWOPI*u2));
		else if (u2 >= 0.75f) phi = atan(alphaV / alphaU*tan(TWOPI*u2)) + TWOPI;
		else phi = atan(alphaV / alphaU*tan(TWOPI*u2)) + PI;
		float sinphi = sin(phi), cosphi = cos(phi);
		float sinphi2 = sinphi * sinphi;
		float cosphi2 = 1.0f - sinphi2;
		float inverseA = 1.0f / (cosphi2 / (alphaU*alphaU) + sinphi2 / (alphaV*alphaV));
		float theta = atan(sqrt(inverseA * u1 / (1.0f - u1)));
		float sintheta = sin(theta), costheta = cos(theta);
		return{
			sintheta*cosphi,
			costheta,
			sintheta*sinphi
		};
	}
}

__device__ inline float3 SchlickFresnel(float3 specular, float costheta){
	float3 rs = specular;
	float c = 1.f - costheta;
	return rs + c*c*c*c*c *(make_float3(1.f, 1.f, 1.f) - rs);
}

__device__ inline float PowerHeuristic(int nf, float fPdf, int ng, float gPdf) {
	float f = nf * fPdf, g = ng * gPdf;
	return (f * f) / (f * f + g * g);
}

//����Դ���ʱ�����ʹ�ö��ַ�����
__device__ int LookUpLightDistribution(float u, float& pdf){
	for (int i = 0; i < kernel_light_distribution_size; ++i){
		float s = kernel_light_distribution[i];
		float e = kernel_light_distribution[i + 1];
		if (u >= s && u <= e){
			pdf = e - s;
			return i;
		}
	}
}

__device__ inline float PdfFromLightDistribution(int idx){
	return kernel_light_distribution[idx + 1] - kernel_light_distribution[idx];
}

__device__ inline void GammaCorrection(float3& in){
	float one_over_gamma = 1.f / 2.2f;
	float exposure = 1.41421356f;

	//pow(x,y) ���ڲ�ʵ����expf(y*log(x)) ����x��Ҫ����0
	in = fmaxf(in, make_float3(1e-5, 1e-5, 1e-5));

	in.x = __powf(in.x*exposure, one_over_gamma);
	in.y = __powf(in.y*exposure, one_over_gamma);
	in.z = __powf(in.z*exposure, one_over_gamma);
}

__device__ inline void FilmicTonemapping(float3& in){
	float3 c = in - make_float3(0.004f, 0.004f, 0.004f);
	c = (c*(6.2f*c + 0.5f)) / (c*(6.2f*c + 1.7f) + 0.06f);
	c = Clamp(c, 0.f, 1.f);
	in = c;
}

__device__ inline float Luminance(const float3& c){
	return dot(c, { 0.212671f, 0.715160f, 0.072169f });
}

__device__ inline bool SameHemiSphere(float3& in, float3& out, float3& nor){
	return dot(in, nor)*dot(out, nor) > 0 ? true : false;
}

__device__ bool Intersect(Ray& ray, Intersection* isect){
	int stack[64];
	int* stack_top = stack;
	int* stack_bottom = stack;

	bool ret = false;
	int node_idx = 0;
	do{
		LinearBVHNode node = kernel_linear[node_idx];
		bool intersect = node.bbox.Intersect(ray);
		if (intersect){
			if (!node.is_leaf){
				*stack_top++ = node.second_child_offset;
				*stack_top++ = node_idx + 1;
			}
			else{
				for (int i = node.start; i <= node.end; ++i){
					Primitive prim = kernel_primitives[i];

					if (prim.type == GT_TRIANGLE){
						if (prim.triangle.Intersect(ray, isect))
							ret = true;
					}
					else{
						if (prim.line.Intersect(ray, isect))
							ret = true;
					}
				}
			}
		}

		if (stack_top == stack_bottom)
			break;
		node_idx = *--stack_top;
	} while (true);

	return ret;
}

__device__ bool IntersectP(Ray& ray){
	int stack[64];
	int* stack_top = stack;
	int* stack_bottom = stack;

	int node_idx = 0;
	do{
		LinearBVHNode node = kernel_linear[node_idx];
		bool intersect = node.bbox.Intersect(ray);
		if (intersect){
			if (!node.is_leaf){
				*stack_top++ = node.second_child_offset;
				*stack_top++ = node_idx + 1;
			}
			else{
				for (int i = node.start; i <= node.end; ++i){
					Primitive prim = kernel_primitives[i];
					if (prim.type == GT_TRIANGLE){
						if (prim.triangle.Intersect(ray, nullptr))
							return true;
					}
					else{
						if (prim.line.Intersect(ray, nullptr))
							return true;
					}
				}
			}
		}

		if (stack_top == stack_bottom)
			break;
		node_idx = *--stack_top;
	} while (true);

	return false;
}

__device__ inline float4 getTexel(Material material, int w, int h, int2 uv){
	float inv = 1.f / 255.f;

	int x = uv.x, y = uv.y;
	float rx = x - (x / w)*w;
	float ry = y - (y / h)*h;
	x = (rx < 0) ? rx + w : rx;
	y = (ry < 0) ? ry + h : ry;
	if (x < 0) x = 0;
	if (x > w - 1) x = w - 1;
	if (y < 0) y = 0;
	if (y > h - 1) y = h - 1;

	uchar4 c = kernel_textures[material.textureIdx][y*w + x];
	return make_float4(c.x*inv, c.y*inv, c.z*inv, c.w * inv);
}

__device__ inline float4 GetTexel(Material material, float2 uv){
	if (material.textureIdx == -1)
		return make_float4(material.diffuse, 1.f);

	int w = kernel_texture_size[material.textureIdx * 2];
	int h = kernel_texture_size[material.textureIdx * 2 + 1];
	float xx = w * uv.x;
	float yy = h * uv.y;
	int x = floor(xx);
	int y = floor(yy);
	float dx = fabs(xx - x);
	float dy = fabs(yy - y);
	float4 c00 = getTexel(material, w, h, make_int2(x, y));
	float4 c10 = getTexel(material, w, h, make_int2(x + 1, y));
	float4 c01 = getTexel(material, w, h, make_int2(x, y + 1));
	float4 c11 = getTexel(material, w, h, make_int2(x + 1, y + 1));
	return (1 - dy)*((1 - dx)*c00 + dx*c10)
		+ dy*((1 - dx)*c01 + dx*c11);
}

__device__ void SampleBSDF(Material material, float3 in, float3 nor, float2 uv, float3 dpdu, float3 u, float3& out, float3& fr, float& pdf){
	switch(material.type){
	case MT_LAMBERTIAN:{
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		out = CosineHemiSphere(u.x, u.y, n, pdf);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		out = ToWorld(out, uu, n, ww);
		fr = make_float3(GetTexel(material, uv)) * ONE_OVER_PI;
		break;
	}
	
	case MT_MIRROR:
		out = 2.f*dot(in, nor)*nor - in;
		fr = material.specular / fabs(dot(out, nor));
		pdf = 1.f;
		break;

	case MT_DIELECTRIC:{
		float3 wi = -in;
		float3 normal = nor;

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wi, normal);
		bool enter = cosi < 0;
		if (!enter){
			float t = ei;
			ei = et;
			et = t;
		}

		float eta = ei / et, cost;
		float sint2 = eta*eta*(1.f - cosi*cosi);
		cost = sqrtf(1.f - sint2 < 0.f ? 0.f : 1.f - sint2);
		float3 rdir = 2.f * dot(-wi, normal) * normal + wi;
		float3 tdir = normalize((wi - normal*cosi)*eta + (enter ? -cost : cost)*normal);
		if (sint2 > 1.f){//total reflection
			out = rdir;
			fr = material.specular / fabs(dot(out, normal));
			pdf = 1.f;
			return;
		}

		float fresnel = DielectricFresnel(fabs(cost), fabs(cosi), et, ei);
		if (u.x > fresnel){//refract
			out = tdir;
			fr = material.specular*eta*eta / fabs(dot(out, normal)) * (1.f - fresnel);
			pdf = 1.f - fresnel;
		}
		else{//reflect
			out = rdir;
			fr = material.specular / fabs(dot(out, normal)) * fresnel;
			pdf = fresnel;
		}
		break;
	}

	case MT_ROUGHCONDUCTOR:{
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float3 wh = SampleGGX(material.alphaU, material.alphaV, u.x, u.y);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		wh = ToWorld(wh, uu, n, ww);
		out = 2.f*dot(in, wh)*wh - in;
		if (!SameHemiSphere(in, out, nor)){
			fr = { 0, 0, 0 };
			pdf = 0.f;
			return;
		}

		float cosi = dot(out, wh);
		float3 F = ConductFresnel(fabs(cosi), material.eta, material.k);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);

		fr = material.specular * F * D * G /
			(4.f * fabs(dot(in, n))*fabs(dot(out, n)));
		pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(in, wh)));
		break;
	}

	case MT_SUBSTRATE:{
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;
		if (u.x < 0.5){
			float ux = u.x * 2.f;
			out = CosineHemiSphere(ux, u.y, n, pdf);
			float3 uu = dpdu, ww;
			ww = cross(uu, n);
			out = ToWorld(out, uu, n, ww);
		}
		else{
			float ux = (u.x - 0.5f) * 2.f;
			float3 wh = SampleGGX(material.alphaU, material.alphaV, ux, u.y);
			float3 uu = dpdu, ww;
			ww = cross(uu, n);
			wh = ToWorld(wh, uu, n, ww);
			out = 2.f * dot(wh, in) * wh - in;
		}
		if (!SameHemiSphere(in, out, n)){
			fr = { 0.f, 0.f, 0.f };
			pdf = 0.f;
			return;
		}
		float c0 = fabs(dot(in, n));
		float c1 = fabs(dot(out, n));
		float3 Rd = make_float3(GetTexel(material, uv));
		float3 Rs = material.specular;
		float cons0 = 1 - 0.5f * c0;
		float cons1 = 1 - 0.5f * c1;
		/*if (u.x < 0.5f){
			float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
				(1 - cons0*cons0*cons0*cons0*cons0) *
				(1 - cons1*cons1*cons1*cons1*cons1);
			fr = diffuse;
			pdf = fabs(dot(out, n)) * ONE_OVER_PI*0.5f;
		}
		else{
			float3 wh = normalize(in + out);
			float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
			float3 specular = D /
				(4.f * fabs(dot(out, wh))*Max(c0, c1))*
				SchlickFresnel(Rs, dot(out, wh));

			fr =  specular;
			pdf = 0.5f * (D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));
		}*/
		float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
			(1 - cons0*cons0*cons0*cons0*cons0) *
			(1 - cons1*cons1*cons1*cons1*cons1);
		float3 wh = normalize(in + out);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float3 specular = D /
			(4.f * fabs(dot(out, wh))*Max(c0, c1))*
			SchlickFresnel(Rs, dot(out, wh));

		fr = diffuse + specular;
		pdf = 0.5f * (fabs(dot(out, n)) * ONE_OVER_PI + D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));

		break;
	}

	case MT_ROUGHDIELECTRIC:{
		float3 wi = -in;
		float3 n = nor;
		float3 wh = SampleGGX(material.alphaU, material.alphaV, u.x, u.y);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		wh = ToWorld(wh, uu, n, ww);

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wi, n);
		bool enter = cosi < 0;
		if (!enter){
			float t = ei;
			ei = et;
			et = t;
		}

		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float eta = ei / et, cost;
		cosi = dot(wi, wh);
		float sint2 = eta*eta*(1.f - cosi*cosi);
		cost = sqrtf(1.f - sint2 < 0.f ? 0.f : 1.f - sint2);
		float3 rdir = 2.f * dot(-wi, wh) * wh + wi;
		float3 tdir = normalize((wi - wh*cosi)*eta + (enter ? -cost : cost)*wh);
		if (sint2 > 1.f){//total reflection
			out = rdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = D*fabs(dot(wh, n)) / (4.f*fabs(dot(wh, in)));
			return;
		}

		float fresnel = DielectricFresnel(fabs(cost), fabs(cosi), et, ei);
		if (u.z > fresnel){//refract
			out = tdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			float c = et*dot(out, wh) + ei*dot(in, wh);
			fr = material.specular*et*et * D * G* (1.f - fresnel) * fabs(dot(in,wh)) * fabs(dot(out,wh)) / 
				(fabs(dot(out, n)) * fabs(dot(in, n)) * c*c);
			pdf = (1.f - fresnel) * D*fabs(dot(wh, n))* et*et*fabs(dot(out, wh)) / (c*c);
		}
		else{//reflect
			out = rdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * fresnel * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = D*fabs(dot(wh, n)) / (4.f*fabs(dot(wh, in))) * fresnel;
		}
		break;
	}
	}
}

//__device__ void Fr(Material material, float3 in, float3 out, float3 nor, float2 uv, float3 dpdu, float u, float3& fr, float& pdf){
__device__ void Fr(Material material, float3 in, float3 out, float3 nor, float2 uv, float3 dpdu, float3& fr, float& pdf){
	switch (material.type){
	case MT_LAMBERTIAN:
		if (!SameHemiSphere(in, out, nor)){
			fr = make_float3(0.f, 0.f, 0.f);
			pdf = 0.f;
			return;
		}

		fr = make_float3(GetTexel(material, uv)) * ONE_OVER_PI;
		pdf = fabs(dot(out, nor)) * ONE_OVER_PI;
		break;

	case MT_MIRROR:
		fr = make_float3(0.f, 0.f, 0.f);
		pdf = 0.f;
		break;

	case MT_DIELECTRIC:
		fr = make_float3(0.f, 0.f, 0.f);
		pdf = 0.f;
		break;

	case MT_ROUGHCONDUCTOR:{
		if (!SameHemiSphere(in, out, nor)){
			fr = { 0, 0, 0 };
			pdf = 0;
			return;
		}
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float3 wh = normalize(in + out);
		float cosi = dot(out, wh);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
		float3 F = ConductFresnel(fabs(cosi), material.eta, material.k);
		fr = material.specular * F*D*G /
			(4.f*fabs(dot(in, n))*fabs(dot(out, n)));
		pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(in, wh)));
		break;
	}

	case MT_SUBSTRATE:{
		if (!SameHemiSphere(in, out, nor)){
			fr = { 0, 0, 0 };
			pdf = 0;
			return;
		}

		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float c0 = fabs(dot(in, n));
		float c1 = fabs(dot(out, n));
		float3 Rd = make_float3(GetTexel(material, uv));
		float3 Rs = material.specular;
		float cons0 = 1 - 0.5f * c0;
		float cons1 = 1 - 0.5f * c1;
		float3 wh = normalize(in + out);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		/*if (D < 1e-4 || u < 0.5f){
			float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
				(1 - cons0*cons0*cons0*cons0*cons0) *
				(1 - cons1*cons1*cons1*cons1*cons1);
			fr = diffuse;
			pdf = 0.5f*fabs(dot(out, n)) * ONE_OVER_PI;
		}
		else{
			float3 specular = D /
				(4.f * fabs(dot(out, wh))*Max(c0, c1))*
				SchlickFresnel(Rs, dot(out, wh));

			fr =  specular;
			pdf = 0.5f * (D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));
		}*/
		float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
			(1 - cons0*cons0*cons0*cons0*cons0) *
			(1 - cons1*cons1*cons1*cons1*cons1);
		float3 specular = D /
			(4.f * fabs(dot(out, wh))*Max(c0, c1))*
			SchlickFresnel(Rs, dot(out, wh));
		fr = diffuse + specular;
		pdf = 0.5f*(fabs(dot(out, n)) * ONE_OVER_PI + D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));
		break;
	}
					  
	case MT_ROUGHDIELECTRIC:{
		float3 wi = -in;
		float3 n = nor;
		bool reflect = dot(in, n)*dot(out, n)>0;

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wi, n);
		bool enter = cosi < 0;
		if (!enter){
			float t = ei;
			ei = et;
			et = t;
		}

		float3 wh = normalize(-(ei*in + et*out));
		float eta = ei / et, cost;
		cosi = dot(wi, wh);
		float sint2 = eta*eta*(1.f - cosi*cosi);
		cost = sqrtf(1.f - sint2 < 0.f ? 0.f : 1.f - sint2);
		float fresnel = DielectricFresnel(fabs(cost), fabs(cosi), et, ei);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		if (!reflect){//refract
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			float c = et*dot(out, wh) + ei*dot(in, wh);
			fr = material.specular*et*et * D * G* (1.f - fresnel) * fabs(dot(in, wh)) * fabs(dot(out, wh)) /
				(fabs(dot(out, n)) * fabs(dot(in, n)) * c*c);
			pdf = (1.f - fresnel) * D*fabs(dot(wh, n))* et*et*fabs(dot(out, wh)) / (c*c);
		}
		else{
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * fresnel * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = fresnel * D*fabs(dot(wh, n)) / (4.f*fabs(dot(wh, in)));

		}
		break;
	}
	}
}

__global__ void Tracing(int iter, int maxDepth){
	unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned pixel = x + y*blockDim.x*gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	hiprandState cudaRNG;
	hiprand_init(WangHash(iter) + threadIndex, 0, 0, &cudaRNG);

	//start
	float offsetx = hiprand_uniform(&cudaRNG) - 0.5f;
	float offsety = hiprand_uniform(&cudaRNG) - 0.5f;
	float unuse;
	float2 aperture = UniformDisk(hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_epsilon;

	float3 Li = make_float3(0.f, 0.f, 0.f);
	float3 beta = make_float3(1.f, 1.f, 1.f);
	Ray r = ray;
	Intersection isect;
	bool specular = false;
	for (int bounces = 0; bounces < maxDepth; ++bounces){
		if (!Intersect(r, &isect)){
			//infinity light
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.nor;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material material = kernel_materials[isect.matIdx];
		if (bounces == 0 || specular){
			if (isect.lightIdx != -1){
				Li += beta*kernel_lights[isect.lightIdx].Le(nor, -r.d);
				break;
			}
		}

		if (IsDiffuse(material.type)){
			float3 Ld = make_float3(0.f, 0.f, 0.f);
			float u = hiprand_uniform(&cudaRNG);
			float choicePdf;
			int idx = LookUpLightDistribution(u, choicePdf);
			Area light = kernel_lights[idx];
			float2 u1 = make_float2(hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG));
			float3 radiance, lightNor;
			Ray shadowRay;
			float lightPdf;
			light.SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);

			bool invisible = IntersectP(shadowRay);
			if (!IsBlack(radiance) && !invisible){
				float3 fr;
				float samplePdf;

				//Fr(material, -r.d, shadowRay.d, nor, uv, dpdu, hiprand_uniform(&cudaRNG), fr, samplePdf);
				Fr(material, -r.d, shadowRay.d, nor, uv, dpdu, fr, samplePdf);
				float weight = PowerHeuristic(1, lightPdf * choicePdf, 1, samplePdf);
				Ld += weight*fr*radiance*fabs(dot(nor, shadowRay.d)) / (lightPdf*choicePdf);
			} 

			float3 uniform = make_float3(hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG));
			float3 out, fr;
			float pdf;
			SampleBSDF(material, -r.d, nor, uv, dpdu, uniform, out, fr, pdf);
			if (!(IsBlack(fr) || pdf == 0)){
				Intersection lightIsect;
				Ray lightRay(pos, out, kernel_epsilon);
				if (Intersect(lightRay, &lightIsect)){
					float3 p = lightIsect.pos;
					float3 n = lightIsect.nor;
					float3 radiance = { 0.f, 0.f, 0.f };
					if (lightIsect.lightIdx != -1)
						radiance = kernel_lights[lightIsect.lightIdx].Le(n, -lightRay.d);
					if (!IsBlack(radiance)){
						float pdfA, pdfW;
						kernel_lights[lightIsect.lightIdx].Pdf(Ray(p, -out, kernel_epsilon), n, pdfA, pdfW);
						float choicePdf = PdfFromLightDistribution(lightIsect.lightIdx);
						float lenSquare = dot(p - pos, p - pos);
						float costheta = fabs(dot(n, lightRay.d));
						float lPdf = pdfA * lenSquare / (costheta);
						float weight = PowerHeuristic(1, pdf, 1, lPdf * choicePdf);
						Ld += weight * fr * radiance * fabs(dot(out, nor)) / pdf;
					}
				}
			}

			Li += beta*Ld;
		}

		float3 u = make_float3(hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG), hiprand_uniform(&cudaRNG));
		float3 out, fr;
		float pdf;

		SampleBSDF(material, -r.d, nor, uv, dpdu, u, out, fr, pdf);
		if (IsBlack(fr))
			break;

		beta *= fr*fabs(dot(nor, out)) / pdf;
		specular = !IsDiffuse(material.type);

		if (bounces > 3){
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (hiprand_uniform(&cudaRNG) < illumate)
				break;

			beta /= (1 - illumate);
		}

		r = Ray(pos, out, kernel_epsilon);
	}

	if (!(isnan(Li.x) || isnan(Li.y) || isnan(Li.z)))
		kernel_color[pixel] = Li;
}

__global__ void Output(int iter, float3* output, bool reset, bool filmic){
	unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned pixel = x + y*blockDim.x*gridDim.x;

	if (reset){
		kernel_acc_image[pixel] = { 0, 0, 0 };
	}
	float3 color = kernel_color[pixel];
	kernel_acc_image[pixel] += color;

	color = kernel_acc_image[pixel] / iter;
	if (filmic)
		FilmicTonemapping(color);
	else
		GammaCorrection(color);
	output[pixel] = color;
}

__global__ void InitRender(
	Camera* camera,
	LinearBVHNode* bvh_nodes,
	Primitive* primitives,
	Material* materials,
	Area* lights,
	uchar4** texs,
	float* light_distribution,
	int ld_size,
	int* tex_size,
	float3* image, 
	float3* color,
	float ep,
	int hdr_w, 
	int hdr_h, 
	bool isvalid){
	kernel_camera = camera;
	kernel_linear = bvh_nodes;
	kernel_primitives = primitives;
	kernel_materials = materials;
	kernel_lights = lights;
	kernel_textures = texs;
	kernel_light_distribution = light_distribution;
	kernel_light_distribution_size = ld_size;
	kernel_texture_size = tex_size;
	kernel_acc_image = image;
	kernel_color = color;
	kernel_epsilon = ep;
	kernel_hdr_width = hdr_w;
	kernel_hdr_height = hdr_h;
	kernel_hdr_isvalid = isvalid;
}

void BeginRender(
	Scene& scene,
	BVH& bvh,
	Camera cam,
	unsigned width,
	unsigned height,
	float ep,
	int max_depth, 
	HDRMap& hdrmap){
	int mesh_memory_use = 0;
	int material_memory_use = 0;
	int bvh_memory_use = 0;
	int light_memory_use = 0;
	int texture_memory_use = 0;
	maxDepth = max_depth;
	int num_primitives = bvh.prims.size();
	HANDLE_ERROR(hipMalloc(&dev_camera, sizeof(Camera)));
	HANDLE_ERROR(hipMemcpy(dev_camera, &cam, sizeof(Camera), hipMemcpyHostToDevice));

	if (num_primitives){
		HANDLE_ERROR(hipMalloc(&dev_primitives, num_primitives*sizeof(Primitive)));
		HANDLE_ERROR(hipMemcpy(dev_primitives, &bvh.prims[0], num_primitives*sizeof(Primitive), hipMemcpyHostToDevice));
		mesh_memory_use += num_primitives*sizeof(Primitive);
	}
	if (bvh.total_nodes > 0){
		HANDLE_ERROR(hipMalloc(&dev_bvh_nodes, bvh.total_nodes*sizeof(LinearBVHNode)));
		HANDLE_ERROR(hipMemcpy(dev_bvh_nodes, bvh.linear_root, bvh.total_nodes*sizeof(LinearBVHNode), hipMemcpyHostToDevice));
		bvh_memory_use += bvh.total_nodes*sizeof(LinearBVHNode);
	}

	//copy material
	int num_materials = scene.materials.size();
	HANDLE_ERROR(hipMalloc(&dev_materials, num_materials*sizeof(Material)));
	HANDLE_ERROR(hipMemcpy(dev_materials, &scene.materials[0], num_materials*sizeof(Material), hipMemcpyHostToDevice));
	material_memory_use += num_materials*sizeof(Material);

	int num_lights = scene.lights.size();
	HANDLE_ERROR(hipMalloc(&dev_lights, num_lights*sizeof(Area)));
	HANDLE_ERROR(hipMemcpy(dev_lights, &scene.lights[0], num_lights*sizeof(Area), hipMemcpyHostToDevice));
	light_memory_use+= num_lights*sizeof(Area);

	//copy textures
	if (scene.textures.size()){
		HANDLE_ERROR(hipMalloc(&texture_size, scene.textures.size() * 2 * sizeof(int)));
		vector<int> texSize;
		HANDLE_ERROR(hipMalloc(&dev_textures, scene.textures.size()*sizeof(uchar4*)));
		for (int i = 0; i < scene.textures.size(); ++i){
			Texture tex = scene.textures[i];
			texSize.push_back(tex.width);
			texSize.push_back(tex.height);
			uchar4* t;
			HANDLE_ERROR(hipMalloc(&t, tex.width*tex.height*sizeof(uchar4)));
			texture_memory_use += tex.width*tex.height*sizeof(uchar4);
			HANDLE_ERROR(hipMemcpy(t, &tex.data[0], tex.width*tex.height*sizeof(uchar4), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(&dev_textures[i], &t, sizeof(uchar4*), hipMemcpyHostToDevice));
		}
		HANDLE_ERROR(hipMemcpy(texture_size, &texSize[0], scene.textures.size() * 2 * sizeof(int), hipMemcpyHostToDevice));
	}

	int num_pixel = width*height;
	HANDLE_ERROR(hipMalloc(&dev_image, num_pixel*sizeof(float3)));
	texture_memory_use += num_pixel*sizeof(float3);
	HANDLE_ERROR(hipMalloc(&dev_color, num_pixel*sizeof(float3)));
	texture_memory_use += num_pixel*sizeof(float3);
	if (hdrmap.isvalid){
		HANDLE_ERROR(hipMalloc(&hdr_map, hdrmap.width*hdrmap.height*sizeof(float4)));
		texture_memory_use += num_pixel*sizeof(float4);
		HANDLE_ERROR(hipMemcpy(hdr_map, hdrmap.image, hdrmap.width*hdrmap.height*sizeof(float4), hipMemcpyHostToDevice));
		hdr_texture.filterMode = hipFilterModeLinear;
		hipChannelFormatDesc channel4desc = hipCreateChannelDesc<float4>();
		HANDLE_ERROR(hipBindTexture(NULL, &hdr_texture, hdr_map, &channel4desc, hdrmap.width*hdrmap.height*sizeof(float4)));
	}

	int ld_size = scene.lightDistribution.size();
	HANDLE_ERROR(hipMalloc(&dev_light_distribution, ld_size*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_light_distribution, &scene.lightDistribution[0], ld_size*sizeof(float), hipMemcpyHostToDevice));
	texture_memory_use += ld_size*sizeof(float);
	
	InitRender << <1, 1 >> >(dev_camera, dev_bvh_nodes,
		dev_primitives, dev_materials, dev_lights, dev_textures, dev_light_distribution, ld_size,
		texture_size, dev_image, dev_color, ep, hdrmap.width, hdrmap.height, hdrmap.isvalid);

	HANDLE_ERROR(hipDeviceSynchronize());

	fprintf(stderr, "\n\nMesh video memory use:[%.3fM]\n", (float)mesh_memory_use / (1024 * 1024));
	fprintf(stderr, "Bvh video memory use:[%.3fM]\n", (float)bvh_memory_use / (1024 * 1024));
	fprintf(stderr, "Material video memory use:[%.3fM]\n", (float)material_memory_use / (1024 * 1024));
	fprintf(stderr, "Light video memory use:[%.3fM]\n", (float)light_memory_use / (1024 * 1024));
	fprintf(stderr, "Texture video memory use:[%.2fM]\n", (float)texture_memory_use / (1024 * 1024));
	fprintf(stderr, "Total video memory use:[%.3fM]\n", (float)(mesh_memory_use + bvh_memory_use + material_memory_use + light_memory_use + texture_memory_use) / (1024 * 1024));
}

void EndRender(){
	HANDLE_ERROR(hipFree(dev_primitives));
	HANDLE_ERROR(hipFree(dev_bvh_nodes));

	HANDLE_ERROR(hipFree(dev_image));
	HANDLE_ERROR(hipFree(dev_color));
	HANDLE_ERROR(hipFree(hdr_map));
	HANDLE_ERROR(hipUnbindTexture(hdr_texture));
}

void Render(Scene& scene, unsigned width, unsigned height, Camera* camera, unsigned iter, bool reset, float3* output){
	HANDLE_ERROR(hipMemcpy(dev_camera, camera, sizeof(Camera), hipMemcpyHostToDevice));
	int block_x = 32, block_y = 4;
	dim3 block(block_x, block_y);
	dim3 grid(width / block.x, height / block.y);

	Tracing << <grid, block >> >(iter, maxDepth);

	grid.x = width / block.x;
	grid.y = height / block.y;
	Output << <grid, block >> >(iter, output, reset, camera->filmic);
}
